#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda_decoder_device.h"

//nvprof --analysis-metrics -o  smpp-server-analysis.nvprof%p --profile-child-processes bin/smpp-server console

extern "C" {
#include "smpp_pdu_struct_cuda.h"
}

static uint8_t *pduBuffer_d;
static CudaPduContext *pduContexts_d;
static CudaDecodedContext *decodedPduStructList_d;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
true

) {
if (code != hipSuccess) {
fprintf(stderr,
"\nGPUassert: %s %s %d\n",
hipGetErrorString(code),
        file, line
);
if (abort)
exit(code);
}
}
__shared__ CudaPduContext directPduContext[1024];

__global__ void launchDecode(int nPduContext, CudaPduContext *pduContexts,
                             CudaDecodedContext *decodedPduStructList, uint8_t *pduBuffer) {
    int threadIndex = ((threadIdx.x + threadIdx.y * blockDim.x) + (blockDim.x * blockDim.y) * threadIdx.z) +
                      (blockDim.x * blockDim.y * blockDim.z) *
                      ((gridDim.x * blockIdx.y) + (blockIdx.x) + (gridDim.x * gridDim.y) * blockIdx.z);


//    printf("ThreadIndex - x - %d | y - %d\n", threadIdx.x, threadIdx.y);
//    printf("decoding launched  - Correlation Id %s | Length - %d\n", pduContexts[0].correlationId, pduContexts[0].start);

    int initBatchSize = nPduContext / ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));
    int remainder = nPduContext % ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));

//    int additionalElements = threadIndex < remainder ? 1 : 0;
    int additionalElements = threadIndex < remainder ? 1 : 0;
    int batchSize = initBatchSize + additionalElements;
    if (batchSize > 0) {
        int i;
        for (i = 0; i < batchSize; i++) {
            int index = threadIndex + i * ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));
//            printf("ThreadIndex - 1 - %d | Index - %d\n", threadIndex, index);
            directPduContext[(threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z) +
                             (i)] = pduContexts[index];
        }
        __syncthreads();

        for (i = 0; i < batchSize; i++) {
            int index = threadIndex + i * ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));
            CudaPduContext *cudaPduContext = &directPduContext[
                    (threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z) + i];
            decodeSinglePdu(
                    &directPduContext[(threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z) + i],
                    &decodedPduStructList[index], pduBuffer);
        }
    }
}

void cudaTest() {
    printf("Cuda SMPP decoding completed\n");

}

CudaPduContext *allocatePinnedPduContext(int length) {
    CudaPduContext *pduContexts;
    gpuErrchk(hipHostMalloc((void **) &pduContexts, sizeof(CudaPduContext) * length));
    return pduContexts;
}

void initCudaParameters(uint32_t pduContextSize, uint64_t pduBufferLength) {
    gpuErrchk(hipMalloc((void **) &pduBuffer_d, sizeof(uint8_t) * pduBufferLength));
    gpuErrchk(hipMalloc((void **) &pduContexts_d, sizeof(CudaPduContext) * pduContextSize));
    gpuErrchk(hipMalloc((void **) &decodedPduStructList_d, sizeof(CudaDecodedContext) * pduContextSize));
}

void freePinndedPduContext(int length, CudaPduContext *pduContexts) {
    hipHostFree(pduContexts);
}

void freePinndedDecodedContext(int length, CudaDecodedContext *decodedPduStructList) {
    hipHostFree(decodedPduStructList);
}

void decodeCuda(CudaMetadata cudaMetadata) {

    CudaPduContext *pduContexts = cudaMetadata.cudaPduContexts;
    CudaDecodedContext *decodedPduStructList = cudaMetadata.decodedPduStructList;
    uint8_t *pduBuffer = cudaMetadata.pduBuffer;
    int nPduContext = cudaMetadata.length;
    int pduContextSize = nPduContext * sizeof(CudaPduContext);
    int decodedContextSize = nPduContext * sizeof(CudaDecodedContext);

    hipMemcpy(pduBuffer_d, pduBuffer, sizeof(uint8_t) * cudaMetadata.pduBufferLength, hipMemcpyHostToDevice);
    hipMemcpy(pduContexts_d, pduContexts, sizeof(CudaPduContext) * nPduContext, hipMemcpyHostToDevice);

    CudaDim block = cudaMetadata.blockDim;
//    printf("Cuda Block Dim | x - %d\n", block.x);
    CudaDim grid = cudaMetadata.gridDim;
    dim3 gridDim(grid.x, grid.y, grid.z);
    dim3 blockDim(block.x, block.y, block.z);

    gpuErrchk(hipPeekAtLastError());
    launchDecode << < gridDim, blockDim >> > (nPduContext, pduContexts_d, decodedPduStructList_d, pduBuffer_d);
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();
    gpuErrchk(
            hipMemcpy(decodedPduStructList, decodedPduStructList_d, sizeof(CudaDecodedContext) * nPduContext,
                       hipMemcpyDeviceToHost));
    gpuErrchk(hipGetLastError());
    hipDeviceSynchronize();
//    hipFree(decodedPduStructList_d);
//    hipFree(pduContexts_d);
//    hipFree(pduBuffer_d);

//    printf("Cuda SMPP decoding completed\n");
}
