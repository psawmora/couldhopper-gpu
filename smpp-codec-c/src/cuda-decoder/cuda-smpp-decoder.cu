#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda_decoder_device.h"
#include "smpp_codec.h"

//nvprof --analysis-metrics -o  smpp-server-analysis.nvprof%p --profile-child-processes bin/smpp-server console

extern "C" {
#include "smpp_pdu_struct_cuda.h"
}

void decodeCudaOneCopyEngine(CudaMetadata cudaMetadata);

void decodeCudaMultipleCopyEngine(CudaMetadata cudaMetadata);

static uint8_t *pduBuffer_d;
static CudaPduContext *pduContexts_d;
static CudaDecodedContext *decodedPduStructList_d;
static int copyEngineCount = 1;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
true

) {
if (code != hipSuccess) {
fprintf(stderr,
"\nGPUassert: %s %s %d\n",
hipGetErrorString(code),
        file, line
);
if (abort)
exit(code);
}
}

__shared__ CudaPduContext directPduContext[1024];
__shared__ uint32_t nPduCount;
__shared__ int minPduIndex;


__global__ void launchDecodeLocal(int nPduContext, int startIndex, CudaPduContext *originalPduContexts,
                                  CudaDecodedContext *originalDecodedPduStructList, uint8_t *pduBuffer) {
    int threadIndex = ((threadIdx.x + threadIdx.y * blockDim.x) + (blockDim.x * blockDim.y) * threadIdx.z) +
                      (blockDim.x * blockDim.y * blockDim.z) *
                      ((gridDim.x * blockIdx.y) + (blockIdx.x) + (gridDim.x * gridDim.y) * blockIdx.z);
     //printf("IN - 2 \n");
    CudaPduContext *pduContexts = originalPduContexts + startIndex;
    CudaDecodedContext *decodedPduStructList = originalDecodedPduStructList + startIndex;
    int initBatchSize = nPduContext / ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));
    int remainder = nPduContext % ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));

    int additionalElements = threadIndex < remainder ? 1 : 0;
    int batchSize = initBatchSize + additionalElements;
    if (batchSize > 0) {
/*
        if ((threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z) == 0) {
            printf("pdu count 1 - 0 - %d\n", nPduContext);
        }
*/
//        printf("batch size - %d\n", batchSize);
        int i;
/*
        for (i = 0; i < batchSize; i++) {
            int index = threadIndex + i * ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));
            int localIndex = (threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z) + i;
            CudaPduContext aStruct1 = pduContexts[index];
            CudaPduContext aStruct2 = directPduContext[localIndex];
            aStruct2 = aStruct1;
        }
*/
//        __syncthreads();
        for (i = 0; i < batchSize; i++) {
            int index = threadIndex + i * ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));
/*
            CudaPduContext *cudaPduContext = &directPduContext[
                    (threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z) + i];
*/
            CudaPduContext *cudaPduContext = &pduContexts[index];
            CudaDecodedContext *decodedPduStruct = &decodedPduStructList[index];
            decodeSinglePdu(cudaPduContext, decodedPduStruct, pduBuffer);
        }
    }
}

__global__ void findPacketBoundry(uint8_t *pduBuffer,
                                  uint32_t bufferLength,
                                  CudaDecodedContext *decodedPduStructList,
                                  CudaPduContext *globalDirectPduContext,
                                  int correlationIdLength) {
    uint8_t pattern[8] = {127, 127, 127, 127, 127, 127, 127, 127};
    int threadIndex = ((threadIdx.x + threadIdx.y * blockDim.x) + (blockDim.x * blockDim.y) * threadIdx.z) +
                      (blockDim.x * blockDim.y * blockDim.z) *
                      ((gridDim.x * blockIdx.y) + (blockIdx.x) + (gridDim.x * gridDim.y) * blockIdx.z);

    uint32_t totalThreadCount = (blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z);
    int init20BytesBatchSize = bufferLength / 20;
    int remainder20Bytes = bufferLength % 20;

    int initBatchSize = init20BytesBatchSize / totalThreadCount;
    int remainder = init20BytesBatchSize % totalThreadCount;
    int additionalElements = threadIndex < remainder ? 1 : 0;
    int batchSize = (initBatchSize + additionalElements) * 20 + (threadIndex == 0) * remainder20Bytes;
    int threadBlockIndex = threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z;
    if (threadBlockIndex == 0) {
        nPduCount = 0;
        minPduIndex = 100000000;
    }
    __syncthreads();

    if (batchSize > 0) {
        int startIndex = threadIndex;
        if (threadIndex < remainder && threadIndex > 0) {
            startIndex = threadIndex * (initBatchSize + additionalElements) * 20 + remainder20Bytes;
        }
        if (threadIndex >= remainder && threadIndex > 0) {
            startIndex = ((remainder) * (initBatchSize + 1) * 20 + remainder20Bytes) +
                         (threadIndex - remainder) * initBatchSize * 20;
        }
        int stop = 0;
        int count = 0;
        int patternPosition = 0;
        int numberOfPdu = 0;

        while (!stop) {
            uint8_t valueAtIndex = *(pduBuffer + startIndex + count);
            if (valueAtIndex == pattern[patternPosition]) {
                patternPosition++;
            } else {
                patternPosition = 0;
            }
            if (patternPosition == 8) {
                ++numberOfPdu;
                patternPosition = 0;

                uint64_t breakPoint = (uint64_t) (startIndex + count + 1);
                int index = readUint32WithoutContext(pduBuffer, (uint64_t) breakPoint);
                CudaPduContext *pduContext = &globalDirectPduContext[index];
                breakPoint += 4;

                uint64_t nextReadIndex =
                        readStringByLength(
                                (uint64_t) (breakPoint), pduBuffer, correlationIdLength, pduContext->correlationId);
                breakPoint = nextReadIndex;

                uint32_t pduLength = readUint32WithoutContext(pduBuffer, (uint64_t) breakPoint);
                pduContext->start = (uint32_t) breakPoint;
                pduContext->length = pduLength;
                breakPoint += pduLength - 1;
                atomicMin(&minPduIndex, index);
                /**
                 * 1. Create CudaPduContext with start and length values.
                 * 2. Store it in block shared memory.
                 * 3. After the block is done processing, store them in the global memory.
                 * 4. Call the child kernel to do the actual decoding.
                 */
            }
            count++;
            if ((patternPosition == 0 && count >= batchSize) || (startIndex + count) >= bufferLength) {
                stop = 1;
            }
        }
//        printf("Pdu Count - %d\n", numberOfPdu);
        atomicAdd(&nPduCount, numberOfPdu);
    }

    __syncthreads();
    if (threadBlockIndex == 0) {
        if (nPduCount > 0) {
            int blockSize = nPduCount >= 256 ? 256 / 3 : nPduCount / 3;
            int gridSize = (nPduCount / blockSize) >= 1 ? (nPduCount / blockSize) : 1;
            printf("Pdu Count - 0 - %d - min - %d - Grid - %d and Block - %d\n", nPduCount, minPduIndex,gridSize, blockSize);

            dim3 gridDim(gridSize, 1, 1);
            dim3 blockDim(blockSize, 1, 1);
            launchDecodeLocal << < gridDim, blockDim >> >
                                            (nPduCount, minPduIndex, globalDirectPduContext, decodedPduStructList, pduBuffer);
        }
        hipDeviceSynchronize();
    }
}

__global__ void launchDecode(int nPduContext, CudaPduContext *pduContexts,
                             CudaDecodedContext *decodedPduStructList, uint8_t *pduBuffer) {
    int threadIndex = ((threadIdx.x + threadIdx.y * blockDim.x) + (blockDim.x * blockDim.y) * threadIdx.z) +
                      (blockDim.x * blockDim.y * blockDim.z) *
                      ((gridDim.x * blockIdx.y) + (blockIdx.x) + (gridDim.x * gridDim.y) * blockIdx.z);


//    printf("ThreadIndex - x - %d | y - %d\n", threadIdx.x, threadIdx.y);
//    printf("decoding launched  - Correlation Id %s | Length - %d\n", pduContexts[0].correlationId, pduContexts[0].start);

    int initBatchSize = nPduContext / ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));
    int remainder = nPduContext % ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));

//    int additionalElements = threadIndex < remainder ? 1 : 0;
    int additionalElements = threadIndex < remainder ? 1 : 0;
    int batchSize = initBatchSize + additionalElements;
    if (batchSize > 0) {
        int i;
        for (i = 0; i < batchSize; i++) {
            int index = threadIndex + i * ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));
//            printf("ThreadIndex - 1 - %d | Index - %d\n", threadIndex, index);
            directPduContext[(threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z) + i]
                    = pduContexts[index];
        }
        __syncthreads();

        for (i = 0; i < batchSize; i++) {
            int index = threadIndex + i * ((blockDim.x * blockDim.y * blockDim.z) * (gridDim.x * gridDim.y * gridDim.z));
            CudaPduContext *cudaPduContext = &directPduContext[
                    (threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z) + i];
            decodeSinglePdu(
                    &directPduContext[(threadIdx.x + threadIdx.y * blockDim.x + (blockDim.x * blockDim.y) * threadIdx.z) + i],
                    &decodedPduStructList[index], pduBuffer);
        }
    }
}

void cudaTest() {

}

CudaPduContext *allocatePinnedPduContext(int length) {
    CudaPduContext *pduContexts;
    gpuErrchk(hipHostMalloc((void **) &pduContexts, sizeof(CudaPduContext) * length));
//    pduContexts = (CudaPduContext *)malloc(sizeof(CudaPduContext) * length);
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    copyEngineCount = prop.asyncEngineCount;
    printf("Copy engine count - %d", copyEngineCount);
    return pduContexts;
}

void initCudaParameters(uint32_t pduContextSize, uint64_t pduBufferLength) {
    gpuErrchk(hipMalloc((void **) &pduBuffer_d, sizeof(uint8_t) * pduBufferLength));
    gpuErrchk(hipMalloc((void **) &pduContexts_d, sizeof(CudaPduContext) * pduContextSize));
    gpuErrchk(hipMalloc((void **) &decodedPduStructList_d, sizeof(CudaDecodedContext) * pduContextSize));
}

void freePinndedMemory() {
    hipFree(decodedPduStructList_d);
    hipFree(pduContexts_d);
    hipFree(pduBuffer_d);
}

void freePinndedPduContext(int length, CudaPduContext *pduContexts) {
    hipHostFree(pduContexts);
//    free(pduContexts);
}

void decodeCudaDynamic(CudaMetadata cudaMetadata) {
    CudaDim block = cudaMetadata.blockDim;
    CudaDim grid = cudaMetadata.gridDim;
    dim3 gridDim(grid.x, grid.y, grid.z);
    dim3 blockDim(block.x, block.y, block.z);

    uint8_t *pduBuffer = cudaMetadata.pduBuffer;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    gpuErrchk(hipEventRecord(start, 0));
    hipMemcpy(pduBuffer_d, pduBuffer, sizeof(uint8_t) * cudaMetadata.pduBufferLength, hipMemcpyHostToDevice);

    gpuErrchk(hipPeekAtLastError());
    findPacketBoundry << < gridDim, blockDim >> > (
            pduBuffer_d, cudaMetadata.pduBufferLength, decodedPduStructList_d, pduContexts_d, 15);
    gpuErrchk(hipPeekAtLastError());
    CudaDecodedContext *decodedPduStructList = cudaMetadata.decodedPduStructList;
    gpuErrchk(
            hipMemcpy(decodedPduStructList, decodedPduStructList_d, sizeof(CudaDecodedContext) * cudaMetadata.length,
                       hipMemcpyDeviceToHost));
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipEventRecord(stop, 0));
    gpuErrchk(hipDeviceSynchronize());
    hipEventElapsedTime(&cudaEventRunningTime, start, stop);
}

void decodeCuda(CudaMetadata cudaMetadata) {
    if (copyEngineCount > 1) {
        decodeCudaMultipleCopyEngine(cudaMetadata);
    } else {
        decodeCudaOneCopyEngine(cudaMetadata);
    }
}

void decodeCudaMultipleCopyEngine(CudaMetadata cudaMetadata) {
    CudaPduContext *pduContexts = cudaMetadata.cudaPduContexts;
    CudaDecodedContext *decodedPduStructList = cudaMetadata.decodedPduStructList;
    uint8_t *pduBuffer = cudaMetadata.pduBuffer;
    int nPduContext = cudaMetadata.length;
    int pduContextSize = nPduContext * sizeof(CudaPduContext);
    int decodedContextSize = nPduContext * sizeof(CudaDecodedContext);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int batchSize = nPduContext > cudaStreamCount ? nPduContext / cudaStreamCount : nPduContext;
    int actualStreamCount = nPduContext > cudaStreamCount ? cudaStreamCount : 1;

    hipStream_t streams[actualStreamCount];
    hipError_t results[actualStreamCount];

    CudaDim block = cudaMetadata.blockDim;
    CudaDim grid = cudaMetadata.gridDim;
    int gridx = grid.x < actualStreamCount ? 1 : grid.x / actualStreamCount;
    int gridy = grid.y < actualStreamCount ? 1 : grid.y / actualStreamCount;
    int gridz = grid.z < actualStreamCount ? 1 : grid.z / actualStreamCount;
    dim3 gridDim(gridx, gridy, gridz);
    dim3 blockDim(block.x, block.y, block.z);

    hipEventRecord(start, 0);
    hipMemcpy(pduBuffer_d, pduBuffer, sizeof(uint8_t) * cudaMetadata.pduBufferLength, hipMemcpyHostToDevice);
    int index = 0;
    int i;
    for (i = 0; i < actualStreamCount; i++) {
        results[i] = hipStreamCreate(&(streams[i]));
        int startIndex = index;
        int length = (nPduContext - index) <= batchSize ? (nPduContext - index) : batchSize;
        results[i] =
                hipMemcpyAsync(
                        pduContexts_d + startIndex, pduContexts + startIndex, sizeof(CudaPduContext) * length,
                        hipMemcpyHostToDevice,
                        streams[i]);
        gpuErrchk(hipPeekAtLastError());
        launchDecode << < gridDim, blockDim, 0, streams[i] >> >
                                                (length, pduContexts_d + startIndex, decodedPduStructList_d + startIndex,
                                                        pduBuffer_d);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(
                hipMemcpyAsync(decodedPduStructList + startIndex, decodedPduStructList_d + startIndex,
                                sizeof(CudaDecodedContext) * length,
                                hipMemcpyDeviceToHost, streams[i]));
        gpuErrchk(hipGetLastError());
        index += length;
    }
}

void decodeCudaOneCopyEngine(CudaMetadata cudaMetadata) {

    CudaPduContext *pduContexts = cudaMetadata.cudaPduContexts;
    CudaDecodedContext *decodedPduStructList = cudaMetadata.decodedPduStructList;
    uint8_t *pduBuffer = cudaMetadata.pduBuffer;
    int nPduContext = cudaMetadata.length;
    int pduContextSize = nPduContext * sizeof(CudaPduContext);
    int decodedContextSize = nPduContext * sizeof(CudaDecodedContext);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipStream_t streams[cudaStreamCount];
    hipError_t results[cudaStreamCount];

    hipEventRecord(start, 0);
    hipMemcpy(pduBuffer_d, pduBuffer, sizeof(uint8_t) * cudaMetadata.pduBufferLength, hipMemcpyHostToDevice);
    int streamCount = 0;
    int batchSize = nPduContext > cudaStreamCount ? nPduContext / cudaStreamCount : nPduContext;
    int index = 0;
    while (index < nPduContext) {
        results[streamCount] = hipStreamCreate(&(streams[streamCount]));
        int startIndex = index;
        int length = (nPduContext - index) <= batchSize ? (nPduContext - index) : batchSize;
        index += length;
        results[streamCount] =
                hipMemcpyAsync(
                        pduContexts_d + startIndex, pduContexts + startIndex, sizeof(CudaPduContext) * length,
                        hipMemcpyHostToDevice,
                        streams[streamCount]);
        streamCount++;

    }


    CudaDim block = cudaMetadata.blockDim;
    CudaDim grid = cudaMetadata.gridDim;
    int gridx = grid.x < streamCount ? 1 : grid.x / streamCount;
    int gridy = grid.y < streamCount ? 1 : grid.y / streamCount;
    int gridz = grid.z < streamCount ? 1 : grid.z / streamCount;
    dim3 gridDim(gridx, gridy, gridz);
    dim3 blockDim(block.x, block.y, block.z);
    index = 0;
    int i = 0;
    while (index < nPduContext) {
        int startIndex = index;
        int length = (nPduContext - index) <= batchSize ? (nPduContext - index) : batchSize;
        index += length;
        gpuErrchk(hipPeekAtLastError());
        launchDecode << < gridDim, blockDim, 0, streams[i] >> >
                                                (length, pduContexts_d + startIndex, decodedPduStructList_d + startIndex,
                                                        pduBuffer_d);
        gpuErrchk(hipPeekAtLastError());
        i++;
    }

    index = 0;
    i = 0;
    while (index < nPduContext) {
        int startIndex = index;
        int length = (nPduContext - index) <= batchSize ? (nPduContext - index) : batchSize;
        index += length;
        gpuErrchk(
                hipMemcpyAsync(decodedPduStructList + startIndex, decodedPduStructList_d + startIndex,
                                sizeof(CudaDecodedContext) * length,
                                hipMemcpyDeviceToHost, streams[i]));
        gpuErrchk(hipGetLastError());
        i++;
    }

    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&cudaEventRunningTime, start, stop);
    for (i = 0; i < streamCount; i++) {
        results[i] = hipStreamDestroy(streams[i]);
    }
}